#include "hip/hip_runtime.h"
#include "matmul.cuh"

#include <hip/hip_runtime.h>
#include ""

#include <chrono>
#include <cmath>
#include <chrono>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_ROW       4
#define BLOCK_COL       8
//gridDim = (n)
//blockDim = (4x8)
//A = convert matrix
//dB = data to convert
//dC = converted data
__global__ void matMul44(float* A, float* dB, float* dC, int dataNum){

    __shared__ float dA[16];

    // memory copy : global memory -> shared memory
    dA[threadIdx.y*BLOCK_ROW + threadIdx.x] = A[threadIdx.y*BLOCK_ROW + threadIdx.x];
    __syncthreads();

    int row = threadIdx.x;
    int col = blockDim.y * blockIdx.x + threadIdx.y;

    if (row >= 4 || col >= dataNum)
	{
		return;
	}

    int Aidx = row * BLOCK_ROW;
    int Bidx = blockDim.x * blockDim.y * blockIdx.x + threadIdx.y * BLOCK_ROW;
    int Cidx = col  * BLOCK_ROW + row;


    float sum = 0;
    for(int i=0; i<BLOCK_ROW; i++){
        sum += dA[Aidx + i] * dB[Bidx + i];
    }
    dC[Cidx] = sum;
    
}

void matMul44Wrapper(float* A, float* B , float* C, int dataNum){
    float *dA, *dB, *dC;
    std::chrono::system_clock::time_point alloc_start = std::chrono::system_clock::now();
    hipMalloc(&dA, BLOCK_ROW*BLOCK_COL*sizeof(float));
    hipMemset(dA, 0, BLOCK_ROW*BLOCK_COL*sizeof(float));
    hipMalloc(&dB, BLOCK_ROW*dataNum*sizeof(float));
    hipMemset(dB, 0, BLOCK_ROW*dataNum*sizeof(float));
    hipMalloc(&dC, BLOCK_ROW*dataNum*sizeof(float));
    hipMemset(dC, 0, BLOCK_ROW*dataNum*sizeof(float));
    std::chrono::duration<double> timeGpuMalloc = std::chrono::system_clock::now() - alloc_start;
    printf("gpu malloc elapsed : %lf(ms)\n", timeGpuMalloc * 1000);

    std::chrono::system_clock::time_point cpy_start = std::chrono::system_clock::now();
    hipMemcpy(dA, A, BLOCK_ROW*BLOCK_COL*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, BLOCK_ROW*dataNum*sizeof(float), hipMemcpyHostToDevice);
    std::chrono::duration<double> timeGpuCpy = std::chrono::system_clock::now() - cpy_start;
    printf("gpu copy elapsed : %lf(ms)\n", timeGpuCpy * 1000);

    dim3 blockDim(BLOCK_ROW, BLOCK_COL);
    dim3 gridDim(ceil(static_cast<float>(dataNum) / BLOCK_COL));

    printf("Grid(%d), Block(%d, %d)\n", gridDim.x, blockDim.x, blockDim.y);
    std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
    matMul44 <<<gridDim, blockDim>>> (dA, dB, dC, dataNum);
    std::chrono::duration<double> timeGpuMatmul = std::chrono::system_clock::now() - start;
    printf("gpu matmul elapsed : %lf(ms)\n", timeGpuMatmul * 1000);

    hipDeviceSynchronize();

    hipMemcpy(C, dC, BLOCK_ROW*dataNum*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

}