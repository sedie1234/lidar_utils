#include <hip/hip_runtime.h>

// 벡터 덧셈을 수행하는 CUDA 커널 함수
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}
